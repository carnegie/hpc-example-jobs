#include <stdio.h>
#include <hip/hip_runtime.h>

int main( ) {
        int deviceCount = 0;
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, deviceCount);

        // This function call returns 0 if there are no CUDA capable devices.
        if (deviceCount == 0) {
                printf("There are no available device(s) that support CUDA\n");
        } else {
                printf("Detected %d CUDA Capable device(s)\n", deviceCount);
        }

        printf("device id %d, name %s\n", deviceCount, prop.name);
        printf("number of multi-processors = %d\n",
            prop.multiProcessorCount);
        printf("Total constant memory: %4.2f kb\n",
            prop.totalConstMem/1024.0);
        printf("Memory Clock rate: %.0f Mhz\n",
           prop.memoryClockRate * 1e-3f);
        printf("GPU Max Clock rate: %.0f MHz (%0.2f ""GHz)\n",
           prop.clockRate * 1e-3f, prop.clockRate * 1e-6f);
        printf("Memory Bus Width: %d-bit\n",
           prop.memoryBusWidth);
        printf("Shared memory per block: %4.2f kb\n",
            prop.sharedMemPerBlock/1024.0);
        printf("Total registers per block: %d\n",
            prop.regsPerBlock);
        printf("Maximum threads per block: %d\n",
            prop.maxThreadsPerBlock);
        printf("Maximum threads per multi-processor: %d\n",
            prop.maxThreadsPerMultiProcessor);
        printf("Maximum number of warps per multi-processor %d\n",
            prop.maxThreadsPerMultiProcessor/32);
        printf("Run time limit on kernels: %s\n",
            prop.kernelExecTimeoutEnabled ? "Yes" : "No");
        printf("Integrated GPU sharing Host Memory: %s\n",
            prop.integrated ? "Yes" : "No");
        printf("Support host page-locked memory mapping: %s\n",
            prop.canMapHostMemory ? "Yes" : "No");
        printf("Alignment requirement for Surfaces: %s\n",
            prop.surfaceAlignment ? "Yes" : "No");
        printf("Device has ECC support: %s\n",
            prop.ECCEnabled ? "Enabled" : "Disabled");
        return 0;
}